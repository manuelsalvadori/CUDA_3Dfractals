#include "hip/hip_runtime.h"
#include <Fract.h>

int main()
{
	int width = WIDTH;
	int height = HEIGHT;

	// window creation and setting
	sf::ContextSettings settings;
	settings.antialiasingLevel = 8;
	sf::RenderWindow window(sf::VideoMode(width, height), "3D fractal", sf::Style::Default, settings);
	window.setVerticalSyncEnabled(true);
	sf::Color background(0, 0, 0, 255);

	sf::Sprite sprite;
	sf::Texture texture;
	sf::Image fractal;
	float3 view = { 0.f, 0.f, -1.f };
	Fract fract(width, height);

	// ClockRate
	int peakClk = 1;
	CHECK(hipDeviceGetAttribute(&peakClk, hipDeviceAttributeClockRate, 0));

	// Events used to compute execution time
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// Host pinned memory allocation
	pixelRegionForStream* imageHost[NUM_STREAMS];
	// Device memory allocation
	pixelRegionForStream* imgDevice[NUM_STREAMS];
	// Create necessary streams
	hipStream_t stream[NUM_STREAMS];


	for (int i = 0; i < NUM_STREAMS; i++) {
		CHECK(hipHostMalloc((pixelRegionForStream**)&imageHost[i], sizeof(pixelRegionForStream)));
		CHECK(hipMalloc((pixelRegionForStream**)&imgDevice[i], sizeof(pixelRegionForStream)));
		CHECK(hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking));
	}

	////Test, setta a bianco
	//for (int i = 0; i < NUM_STREAMS; i++)
	//{
	//	for (int j = 0; j < PIXEL_PER_STREAM; j++)
	//	{
	//		pixel* temp = *imageHost[i];
	//		temp[j].r = 255;
	//		temp[j].g = 0;
	//		temp[j].b = 0;
	//	}
	//}


	int frameCounter = 0;

	// loop
	while (window.isOpen())
	{
		printf("Frame Numero %d\n", frameCounter);
		CHECK(hipEventRecord(start));
		window.clear(background);
		texture.loadFromImage(*fract.generateFractal(view, imgDevice[0], imageHost[0], stream, peakClk));
		sprite.setTexture(texture, true);
		window.draw(sprite);
		CHECK(hipEventRecord(stop));

		CHECK(hipEventSynchronize(start));
		CHECK(hipEventSynchronize(stop));
		float milliseconds = 0;
		CHECK(hipEventElapsedTime(&milliseconds, start, stop));
		printf("Tempo calcolo frame: %fs\n", (milliseconds / 1000));
		printf("--------------\n", (milliseconds / 1000));
		frameCounter++;


		// event handling
		sf::Event event;
		while (window.pollEvent(event))
		{
			if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::Right)
			{
				// Camera movement
				// ...
			}

			if (event.type == sf::Event::Closed)
				window.close();
		}
		window.display();
	}

	// Cleanup
	hipHostFree(imageHost);
	hipFree(imgDevice);
	for (int i = 0; i < 6; i++) {
		CHECK(hipStreamDestroy(stream[i]));
	}

	return 0;
}
