#include "hip/hip_runtime.h"
#include <Fract.h>

int main()
{
	int width = WIDTH;
	int height = HEIGHT;

	// window creation and setting
	sf::ContextSettings settings;
	settings.antialiasingLevel = 8;
	sf::RenderWindow window(sf::VideoMode(width, height), "3D fractal", sf::Style::Default, settings);
	window.setVerticalSyncEnabled(true);
	sf::Color background(0, 0, 0, 255);

	sf::Sprite sprite;
	sf::Texture texture;
	sf::Image fractal;
	float3 view = { 0.f, 0.f, -1.f };
	Fract fract(width, height);

	// Host memory allocation
	pixel* imageHost;
	CHECK(hipHostMalloc((pixel**)&imageHost, sizeof(pixel)*width*height));

	// Device memory allocation
	pixel* imgDevice;
	CHECK(hipMalloc((pixel**)&imgDevice, sizeof(pixel)*width*height));

	//// Costant memory allocation
	//sf::Vector3f upH(0, 1, 0);
	//sf::Vector3f rightH(1, 0, 0);
	//CHECK(hipMemcpyToSymbol(HIP_SYMBOL(upDevice), &upH, sizeof(upH), 0, hipMemcpyHostToDevice));
	//CHECK(hipMemcpyToSymbol(HIP_SYMBOL(rightDevice), &rightH, sizeof(rightH), 0, hipMemcpyHostToDevice));

	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printf("Timeout attivato: %d.\n", devProp.kernelExecTimeoutEnabled);

	// loop
	while (window.isOpen())
	{
		window.clear(background);

		texture.loadFromImage(*fract.generateFractal(view, imgDevice, imageHost));
		sprite.setTexture(texture, true);
		window.draw(sprite);


		// event handling
		sf::Event event;
		while (window.pollEvent(event))
		{
			if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::Down)
			{
				// move camera
			}

			if (event.type == sf::Event::Closed)
				window.close();
		}
		window.display();
	}
	return 0;
}
